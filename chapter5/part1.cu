
#include <hip/hip_runtime.h>
// Kernel de 5.13 modificado (Ejercicio 5.1)
__global__
void kernel513modified(float* X, float *Y, int size) {
  extern __shared__ float partialSum[];
	unsigned int t = threadIdx.x;
	partialSum[t] = 0;
	if (blockIdx.x*blockDim.x + threadIdx.x < size) {
		partialSum[t] = X[blockIdx.x*blockDim.x + threadIdx.x];
	}
	__syncthreads();
	for (unsigned int stride = 1; stride < 2048; stride <<= 2) {
		if (t % (2 * stride) == 0) {
			partialSum[t] += partialSum[t + stride];
		}
		__syncthreads();  // previene a los threads de sobreescribir la memoria compartida si terminan antes
	}
}

// Kernel de 5.15 modificado (Ejercicio 5.1)
__global__
void kernel515modified(float* X, float* Y, int size) {
	extern __shared__ float partialSum[];
	unsigned int t = threadIdx.x;
	partialSum[t] = 0;
	if (blockIdx.x*blockDim.x + threadIdx.x < size) {
		partialSum[t] = X[blockIdx.x*blockDim.x + threadIdx.x];
	}
	__syncthreads();
	for (unsigned int stride = blockDim.x/2; stride > 0; stride >>= 1) {
		if (t < stride) {
			partialSum[t] += partialSum[t + stride];
		}
		__syncthreads();
	}
}
