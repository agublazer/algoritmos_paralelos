#include "hip/hip_runtime.h"
// Kernel de 5.13 modificado (Ejercicio 5.1)
__global__
void kernel513modified(float* X, float *Y, int size) {
  extern __shared__ float partialSum[];
	unsigned int t = threadIdx.x;
	partialSum[t] = 0;
	if (blockIdx.x*blockDim.x + threadIdx.x < size) {
		partialSum[t] = X[blockIdx.x*blockDim.x + threadIdx.x];
	}
	__syncthreads();
	for (unsigned int stride = 1; stride < 2048; stride <<= 2) {
		if (t % (2 * stride) == 0) {
			partialSum[t] += partialSum[t + stride];
		}
		__syncthreads();  // previene a los threads de sobreescribir la memoria compartida si terminan antes
	}
}

// Kernel de 5.15 modificado (Ejercicio 5.1)
__global__
void kernel515modified(float* X, float* Y, int size) {
	extern __shared__ float partialSum[];
	unsigned int t = threadIdx.x;
	partialSum[t] = 0;
	if (blockIdx.x*blockDim.x + threadIdx.x < size) {
		partialSum[t] = X[blockIdx.x*blockDim.x + threadIdx.x];
	}
	__syncthreads();
	for (unsigned int stride = blockDim.x/2; stride > 0; stride >>= 1) {
		if (t < stride) {
			partialSum[t] += partialSum[t + stride];
		}
		__syncthreads();
	}
}

// Kernel basado en 5.1 (Ejercicio 3)
__global__
void 51basedkernel(float* X, float* Y, int size) {
	extern __shared__ float partialSum[];
	unsigned int t = threadIdx.x;
	unsigned int i = ;
	partialSum[t] = 0;
	if (blockIdx.x*(blockDim.x*2) + threadIdx.x < size) {
		partialSum[t] = X[blockIdx.x*(blockDim.x*2) + threadIdx.x];
    partialSum[t] += X[blockIdx.x*(blockDim.x*2) + threadIdx.x + blockDim.x];
	}
	__syncthreads();
	for (unsigned int stride = blockDim.x/2; stride > 0; stride >>= 1) {
		if (t < stride) {
			partialSum[t] += partialSum[t + stride];
		}
		__syncthreads();
	}
}

