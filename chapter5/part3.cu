#include "hip/hip_runtime.h"
// Kernel basado en 5.1 (Ejercicio 3)
__global__
void 51basedkernel(float* X, float* Y, int size) {
	extern __shared__ float partialSum[]; 
	unsigned int t = threadIdx.x;
	partialSum[t] = 0;
	if (blockIdx.x*(blockDim.x*2) + threadIdx.x < size) { // (1)
		partialSum[t] = X[blockIdx.x*(blockDim.x*2) + threadIdx.x];
    		partialSum[t] += X[blockIdx.x*(blockDim.x*2) + threadIdx.x + blockDim.x];
	}
	__syncthreads();
	for (unsigned int stride = blockDim.x/2; stride > 0; stride >>= 1) { // (2)
		if (t < stride) {
			partialSum[t] += partialSum[t + stride]; // (3)
		}
		__syncthreads();
	}
}
