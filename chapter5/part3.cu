#include "hip/hip_runtime.h"
// Kernel basado en 5.1 (Ejercicio 3)
__global__
void 51basedkernel(float* X, float* Y, int size) {
	extern __shared__ float partialSum[];
	unsigned int t = threadIdx.x;
	unsigned int i = ;
	partialSum[t] = 0;
	if (blockIdx.x*(blockDim.x*2) + threadIdx.x < size) {
		partialSum[t] = X[blockIdx.x*(blockDim.x*2) + threadIdx.x];
    partialSum[t] += X[blockIdx.x*(blockDim.x*2) + threadIdx.x + blockDim.x];
	}
	__syncthreads();
	for (unsigned int stride = blockDim.x/2; stride > 0; stride >>= 1) {
		if (t < stride) {
			partialSum[t] += partialSum[t + stride];
		}
		__syncthreads();
	}
}
