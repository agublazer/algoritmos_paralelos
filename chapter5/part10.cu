#include "hip/hip_runtime.h"
//Ejercicio 10
__global__
void figure517kernel(float* A, float* B, float* C, int Width) {
	__shared__ float Mds[TILE_WIDTH][WIDTH];
	__shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

	int bx = blockIdx.x;  int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;

	int Row = by * TILE_WIDTH + ty;
	int Col = bx * TILE_WIDTH + tx;

	float Pvalue = 0;
	for (int ph = 0; ph < ceil(Width / (float)TILE_WIDTH); ++ph) {
		if ((Row < Width) && ((ph*WIDTH + tx) < Width))
			Mds[ty][tx] = A[Row*Width + ph*WIDTH + tx];
		if (((ph*TILE_WIDTH + ty) < Width) && (Col < Width))
			Nds[ty][tx] = B[(ph*TILE_WIDTH + ty)*Width + Col];
		
    		__syncthreads(); //Nos aseguramos que todos los threads cargan los datos antes de continuar

		for (int k = 0; k < TILE_WIDTH; ++k) {
			Pvalue += Mds[ty][k] * Nds[k][tx];
		}
		__syncthreads(); // Nos aseguramos que todos los threads hayan usado los datos antes de jalar nuevos datos
	}
	if ((Row < Width) && (Col < Width))
		C[Row*Width + Col] = Pvalue;
}
