#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdio>
#include <iostream>
#include <chrono>

using namespace std;
// check for errors using cuda runtime api
// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void vecAdd1(float *h_A, float* h_B, float* h_C, int n) {
        for (int i = 0; i < n; i++) {
                h_C[i] = h_A[i] + h_B[i];
        }
}

__global__
void vecAddKernel(float* A, float* B, float* C, int n) {
        int i = blockDim.x*blockIdx.x + threadIdx.x;
        if (i < n) {
                C[i] = A[i] + B[i];
        }
}

void vecAdd2(float* A, float* B, float* C, int n) {
        int size = n * sizeof(float);
        float *d_A, *d_B, *d_C;
        hipMalloc(&d_A, size);
        hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
        hipMalloc(&d_B, size);
        hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

        hipMalloc(&d_C, size);

        vecAddKernel <<< ceil(n/256.0), 256 >>> (d_A, d_B, d_C, n);
        
        hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
}

int main() {
        float *h_A, *h_B, *h_C;
        int n = 10;
        h_A = (float*)malloc(n * sizeof(float));
        h_B = (float*)malloc(n * sizeof(float));
        h_C = (float*)malloc(n * sizeof(float));
        for (int i = 0; i < n; i++) {
                h_A[i] = 10.0;
                h_B[i] = 20.0;
                h_C[i] = 0.0;
        }

	vecAdd2(h_A, h_B, h_C, n);

        for(int i = 0; i<n; i++){
        	printf("%f ", h_C[i]);
        }
	printf("\n");

        free(h_A);
        free(h_B);
        free(h_C);

        return 0;
}
